#include "hip/hip_runtime.h"

// This is not really C++-code but pretty plain C code, but we compile it
// as C++ so we can integrate with CUDA seamlessly.

// If you plan on submitting your solution for the Parallel Sorting Contest,
// please keep the split into main file and kernel file, so we can easily
// insert other data.

#define BLOCKSIZE 1024

__device__
static void exchange(int *i, int *j)
{
	int k;
	k = *i;
	*i = *j;
	*j = k;
}

__global__
void bitonic_block(int *data, int N, int j, int k)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < N) // stop criteria from for(i=0;i<N;i++) CPU part
    {
      // copy paste from CPU part
      int ixj=i^j; // Calculate indexing!
      if ((ixj)>i)
        {
          if ((i&k)==0 && data[i]>data[ixj]) exchange(&data[i],&data[ixj]);
          if ((i&k)!=0 && data[i]<data[ixj]) exchange(&data[i],&data[ixj]);
        }
     }
}

// No, this is not GPU code yet but just a copy of the CPU code, but this
// is where I want to see your GPU code!
void bitonic_gpu(int *data, int N)
{
  // int i,j,k;
  int j,k;
  int size = sizeof(int) * N;
  int* devicedata;
  hipMalloc( (void**)&devicedata, size);
  hipMemcpy(devicedata, data, size, hipMemcpyHostToDevice);

  dim3 dimBlock(BLOCKSIZE,1);
  dim3 dimGrid(N/BLOCKSIZE,1);

  for (k=2;k<=N;k=2*k) // Outer loop, double size for each step
    {
      for (j=k>>1;j>0;j=j>>1) // Inner loop, half size for each step
        {
          bitonic_block<<<dimGrid, dimBlock>>>(devicedata, N, j, k);
          hipDeviceSynchronize();
          /*
          for (i=0;i<N;i++) // Loop over data
            {
              int ixj=i^j; // Calculate indexing!
              if ((ixj)>i)
                {
                  if ((i&k)==0 && data[i]>data[ixj]) exchange(&data[i],&data[ixj]);
                  if ((i&k)!=0 && data[i]<data[ixj]) exchange(&data[i],&data[ixj]);
                }
            }
          */
        }
    }

  hipMemcpy(data, devicedata, size, hipMemcpyDeviceToHost);

  /* Original CPU sort
  for (k=2;k<=N;k=2*k) // Outer loop, double size for each step
  {
    for (j=k>>1;j>0;j=j>>1) // Inner loop, half size for each step
    {
      for (i=0;i<N;i++) // Loop over data
      {
        int ixj=i^j; // Calculate indexing!
        if ((ixj)>i)
        {
          if ((i&k)==0 && data[i]>data[ixj]) exchange(&data[i],&data[ixj]);
          if ((i&k)!=0 && data[i]<data[ixj]) exchange(&data[i],&data[ixj]);
        }
      }
    }
  }
  */
}
